#include "hip/hip_runtime.h"
// cudanoise
// Library of common 3D noise functions for CUDA kernels

#define N 512
#define WORLDSIZE N * N

#include <hip/hip_runtime.h>
#include "cudanoise.cuh"

__device__ unsigned int hash(unsigned int seed)
{
	seed = (seed + 0x7ed55d16) + (seed << 12);
	seed = (seed ^ 0xc761c23c) ^ (seed >> 19);
	seed = (seed + 0x165667b1) + (seed << 5);
	seed = (seed + 0xd3a2646c) ^ (seed << 9);
	seed = (seed + 0xfd7046c5) + (seed << 3);
	seed = (seed ^ 0xb55a4f09) ^ (seed >> 16);

	return seed;
}

__device__ int randomIntRange(int min, int max, int seed)
{
	int base = hash(seed);
	base = base % (1 + max - min) + min;

	return base;
}

__device__ float randomFloat(unsigned int seed)
{
	unsigned int noiseVal = hash(seed);

	return ((float)noiseVal / (float)0xffffffff);
}

__device__ float clamp(float val, float min, float max)
{
	if (val < 0.0f)
		return 0.0f;
	else if (val > 1.0f)
		return 1.0f;

	return val;
}

__device__ float mapToSigned(float input)
{
	return input * 2.0f - 1.0f;
}

__device__ float mapToUnsigned(float input)
{
	return input * 0.5f + 0.5f;
}

// Random float for a grid coordinate [-1, 1]
__device__ float randomGrid(int x, int y, int z, int seed = 0)
{
	return mapToSigned(randomFloat((unsigned int)(x * 1723 + y * 93241 + z * 149812 + 3824 + seed)));
}

// Random unsigned int for a grid coordinate [0, MAXUINT]
__device__ unsigned int randomIntGrid(int x, int y, int z, int seed = 0)
{
	return hash((unsigned int)(x * 1723 + y * 93241 + z * 149812 + 3824 + seed));
}

__device__ float3 vectorNoise(int x, int y, int z)
{
	return make_float3(randomFloat(x * 8231 + y * 34612 + z * 11836 + 19283) * 2.0f - 1.0f,
		   			   randomFloat(x * 1171 + y * 9234 + z * 992903 + 1466) * 2.0f - 1.0f,
					   0.0f);
}

__device__ float3 scaleVector(float3 v, float factor)
{
	return make_float3(v.x * factor, v.y * factor, v.z * factor);
}

__device__ float3 addVectors(float3 v, float3 w)
{
	return make_float3(v.x + w.x, v.y + w.y, v.z + w.z);
}

__device__ float dotProduct(float3 u, float3 v)
{
	return (u.x * v.x + u.y * v.y + u.z * v.z);
}

// Helper functions for noise noise

__device__ float lerp(float a, float b, float ratio)
{
	return a * (1.0f - ratio) + b * ratio;
}

__device__ float cubic(float p0, float p1, float p2, float p3, float x)
{
	return p1 + 0.5 * x * (p2 - p0 + x * (2.0 * p0 - 5.0 * p1 + 4.0 * p2 - p3 + x * (3.0 * (p1 - p2) + p3 - p0)));
}

__device__ float grad(int hash, float x, float y, float z)
{
	switch (hash & 0xF)
	{
	case 0x0: return  x + y;
	case 0x1: return -x + y;
	case 0x2: return  x - y;
	case 0x3: return -x - y;
	case 0x4: return  x + z;
	case 0x5: return -x + z;
	case 0x6: return  x - z;
	case 0x7: return -x - z;
	case 0x8: return  y + z;
	case 0x9: return -y + z;
	case 0xA: return  y - z;
	case 0xB: return -y - z;
	case 0xC: return  y + x;
	case 0xD: return -y + z;
	case 0xE: return  y - x;
	case 0xF: return -y - z;
	default: return 0; // never happens
	}
}

__device__ float fade(float t)
{
	// Fade function as defined by Ken Perlin.  This eases coordinate values
	// so that they will ease towards integral values.  This ends up smoothing
	// the final output.
	return t * t * t * (t * (t * 6.0f - 15.0f) + 10.0f);         // 6t^5 - 15t^4 + 10t^3
}

// Noise functions

__device__ float checker(float3 pos, float scale, int seed)
{
	int ix = (int)(pos.x * scale);
	int iy = (int)(pos.y * scale);
	int iz = (int)(pos.z * scale);

	if ((ix + iy + iz) % 2 == 0)
		return 1.0f;

	return 0.0f;
}

__device__ float spots(float3 pos, float scale, int seed, float size, int minNum, int maxNum, float jitter, profileShape shape)
{
	if (size < EPSILON)
		return 0.0f;

	int ix = (int)(pos.x * scale);
	int iy = (int)(pos.y * scale);
	int iz = (int)(pos.z * scale);

	float u = pos.x - (float)ix;
	float v = pos.y - (float)iy;
	float w = pos.z - (float)iz;

	float val = -1.0f;

	// We need to traverse the entire 3x3x3 neighborhood in case there are spots in neighbors near the edges of the cell
	for (int x = -1; x < 2; x++)
	{
		for (int y = -1; y < 2; y++)
		{
			for (int z = -1; z < 2; z++)
			{


				int numSpots = randomIntRange(minNum, maxNum, seed + (ix + x) * 823746 + (iy + y) * 12306 + (iz + z) * 823452 + 3234874);
				numSpots = 1;

				for (int i = 0; i < numSpots; i++)
				{
					float distU = u - x - (randomFloat(seed + (ix + x) * 23784 + (iy + y) * 9183 + (iz + z) * 23874 + 334 * i + 27432) * jitter - jitter / 2.0f);
					float distV = v - y - (randomFloat(seed + (ix + x) * 12743 + (iy + y) * 45191 + (iz + z) * 144421 + 2934 * i + 76671) * jitter - jitter / 2.0f);
					float distW = w - z - (randomFloat(seed + (ix + x) * 82734 + (iy + y) * 900213 + (iz + z) * 443241 + 18237 * i + 199823) * jitter - jitter / 2.0f);

					float distanceSq = distU * distU + distV * distV + distW * distW;

					switch (shape)
					{
					case(CUDANOISE_STEP):
						if (distanceSq < size)
							val = fmaxf(val, 1.0f);
						else
							val = -1.0f;
						break;
					case(CUDANOISE_LINEAR):
						val = fmaxf(val, sqrtf(distanceSq));
						break;
					case(CUDANOISE_QUADRATIC):
						val = fmaxf(val, 1.0f - clamp(distanceSq, 0.0f, size) / size);
						break;
					}
				}
			}
		}
	}

	return val;
}

__device__ float spotsOld(float3 pos, float scale, int seed, float size, int minNum, int maxNum, float jitter, profileShape shape)
{
	if (size < EPSILON)
		return 0.0f;

	int ix = (int)(pos.x * scale);
	int iy = (int)(pos.y * scale);
	int iz = (int)(pos.z * scale);

	float u = pos.x - (float)ix;
	float v = pos.y - (float)iy;
	float w = pos.z - (float)iz;

	int numSpots = randomIntRange(minNum, maxNum, seed + ix * 823746 + iy * 12306 + iz * 823452 + 3234874);

	float val = -1.0f;

	for (int i = 0; i < numSpots; i++)
	{
		float distU = 0.5f - u + randomFloat(seed + ix * 23784 + iy * 9183 + iz * 23874 + 334 * i + 27432) * jitter - jitter / 2.0f;
		float distV = 0.5f - v + randomFloat(seed + ix * 12743 + iy * 45191 + iz * 144421 + 2934 * i + 76671) * jitter - jitter / 2.0f;
		float distW = 0.5f - w + randomFloat(seed + ix * 82734 + iy * 900213 + iz * 443241 + 18237 * i + 199823) * jitter - jitter / 2.0f;
		float distanceSq = distU * distU + distV * distV + distW * distW;

		switch (shape)
		{
		case(CUDANOISE_STEP):
			if (distanceSq < size)
				val = fmaxf(val, 1.0f);
			else
				val = -1.0f;
			break;
		case(CUDANOISE_LINEAR):
			val = fmaxf(val, sqrtf(distanceSq));
			break;
		case(CUDANOISE_QUADRATIC):
			val = fmaxf(val, 1.0f - clamp(distanceSq, 0.0f, size) / size);
			break;
		}
	}

	return val;
}

__device__ float tricubic(int x, int y, int z, float u, float v, float w)
{
	// interpolate along x first
	float x00 = cubic(randomGrid(x - 1, y - 1, z - 1), randomGrid(x, y - 1, z - 1), randomGrid(x + 1, y - 1, z - 1), randomGrid(x + 2, y - 1, z - 1), u);
	float x01 = cubic(randomGrid(x - 1, y - 1, z), randomGrid(x, y - 1, z), randomGrid(x + 1, y - 1, z), randomGrid(x + 2, y - 1, z), u);
	float x02 = cubic(randomGrid(x - 1, y - 1, z + 1), randomGrid(x, y - 1, z + 1), randomGrid(x + 1, y - 1, z + 1), randomGrid(x + 2, y - 1, z + 1), u);
	float x03 = cubic(randomGrid(x - 1, y - 1, z + 2), randomGrid(x, y - 1, z + 2), randomGrid(x + 1, y - 1, z + 2), randomGrid(x + 2, y - 1, z + 2), u);

	float x10 = cubic(randomGrid(x - 1, y, z - 1), randomGrid(x, y, z - 1), randomGrid(x + 1, y, z - 1), randomGrid(x + 2, y, z - 1), u);
	float x11 = cubic(randomGrid(x - 1, y, z), randomGrid(x, y, z), randomGrid(x + 1, y, z), randomGrid(x + 2, y, z), u);
	float x12 = cubic(randomGrid(x - 1, y, z + 1), randomGrid(x, y, z + 1), randomGrid(x + 1, y, z + 1), randomGrid(x + 2, y, z + 1), u);
	float x13 = cubic(randomGrid(x - 1, y, z + 2), randomGrid(x, y, z + 2), randomGrid(x + 1, y, z + 2), randomGrid(x + 2, y, z + 2), u);

	float x20 = cubic(randomGrid(x - 1, y + 1, z - 1), randomGrid(x, y + 1, z - 1), randomGrid(x + 1, y + 1, z - 1), randomGrid(x + 2, y + 1, z - 1), u);
	float x21 = cubic(randomGrid(x - 1, y + 1, z), randomGrid(x, y + 1, z), randomGrid(x + 1, y + 1, z), randomGrid(x + 2, y + 1, z), u);
	float x22 = cubic(randomGrid(x - 1, y + 1, z + 1), randomGrid(x, y + 1, z + 1), randomGrid(x + 1, y + 1, z + 1), randomGrid(x + 2, y + 1, z + 1), u);
	float x23 = cubic(randomGrid(x - 1, y + 1, z + 2), randomGrid(x, y + 1, z + 2), randomGrid(x + 1, y + 1, z + 2), randomGrid(x + 2, y + 1, z + 2), u);

	float x30 = cubic(randomGrid(x - 1, y + 2, z - 1), randomGrid(x, y + 2, z - 1), randomGrid(x + 1, y + 2, z - 1), randomGrid(x + 2, y + 2, z - 1), u);
	float x31 = cubic(randomGrid(x - 1, y + 2, z), randomGrid(x, y + 2, z), randomGrid(x + 1, y + 2, z), randomGrid(x + 2, y + 2, z), u);
	float x32 = cubic(randomGrid(x - 1, y + 2, z + 1), randomGrid(x, y + 2, z + 1), randomGrid(x + 1, y + 2, z + 1), randomGrid(x + 2, y + 2, z + 1), u);
	float x33 = cubic(randomGrid(x - 1, y + 2, z + 2), randomGrid(x, y + 2, z + 2), randomGrid(x + 1, y + 2, z + 2), randomGrid(x + 2, y + 2, z + 2), u);

	// interpolate along y
	float y0 = cubic(x00, x10, x20, x30, v);
	float y1 = cubic(x01, x11, x21, x31, v);
	float y2 = cubic(x02, x12, x22, x32, v);
	float y3 = cubic(x03, x13, x23, x33, v);

	// interpolate along z
	return cubic(y0, y1, y2, y3, w);
}

__device__ float discreteNoise(float3 pos, float scale, int seed)
{
	int ix = (int)(pos.x * scale);
	int iy = (int)(pos.y * scale);
	int iz = (int)(pos.z * scale);

	return randomGrid(ix, iy, iz, seed);
}

__device__ float linearValue(float3 pos, float scale, int seed)
{
	int ix = (int)pos.x;
	int iy = (int)pos.y;
	int iz = (int)pos.z;

	float u = pos.x - ix;
	float v = pos.y - iy;
	float w = pos.z - iz;

	// Corner values
	float a000 = randomGrid(ix, iy, iz, seed);
	float a100 = randomGrid(ix + 1, iy, iz, seed);
	float a010 = randomGrid(ix, iy + 1, iz, seed);
	float a110 = randomGrid(ix + 1, iy + 1, iz, seed);
	float a001 = randomGrid(ix, iy, iz + 1, seed);
	float a101 = randomGrid(ix + 1, iy, iz + 1, seed);
	float a011 = randomGrid(ix, iy + 1, iz + 1, seed);
	float a111 = randomGrid(ix + 1, iy + 1, iz + 1, seed);

	// Linear interpolation
	float x00 = lerp(a000, a100, u);
	float x10 = lerp(a010, a110, u);
	float x01 = lerp(a001, a101, u);
	float x11 = lerp(a011, a111, u);

	float y0 = lerp(x00, x10, v);
	float y1 = lerp(x01, x11, v);

	return lerp(y0, y1, w);
}

__device__ float fadedValue(float3 pos, float scale, int seed)
{
	int ix = (int)(pos.x * scale);
	int iy = (int)(pos.y * scale);
	int iz = (int)(pos.z * scale);

	float u = fade(pos.x - ix);
	float v = fade(pos.y - iy);
	float w = fade(pos.z - iz);

	// Corner values
	float a000 = randomGrid(ix, iy, iz);
	float a100 = randomGrid(ix + 1, iy, iz);
	float a010 = randomGrid(ix, iy + 1, iz);
	float a110 = randomGrid(ix + 1, iy + 1, iz);
	float a001 = randomGrid(ix, iy, iz + 1);
	float a101 = randomGrid(ix + 1, iy, iz + 1);
	float a011 = randomGrid(ix, iy + 1, iz + 1);
	float a111 = randomGrid(ix + 1, iy + 1, iz + 1);

	// Linear interpolation
	float x00 = lerp(a000, a100, u);
	float x10 = lerp(a010, a110, u);
	float x01 = lerp(a001, a101, u);
	float x11 = lerp(a011, a111, u);

	float y0 = lerp(x00, x10, v);
	float y1 = lerp(x01, x11, v);

	return lerp(y0, y1, w) / 2.0f * 1.0f;
}

__device__ float cubicValue(float3 pos, float scale, int seed)
{
	pos.x = pos.x * scale;
	pos.y = pos.y * scale;
	pos.z = pos.z * scale;

	int ix = (int)pos.x;
	int iy = (int)pos.y;
	int iz = (int)pos.z;

	float u = pos.x - ix;
	float v = pos.y - iy;
	float w = pos.z - iz;

	return tricubic(ix, iy, iz, u, v, w);
}

__device__ float perlinNoise(float3 pos, float scale, int seed)
{
	pos.x = pos.x * scale;
	pos.y = pos.y * scale;
	pos.z = pos.z * scale;

	// zero corner integer position
	int ix = (int)floorf(pos.x);
	int iy = (int)floorf(pos.y);
	int iz = (int)floorf(pos.z);

	// current position within unit cube
	pos.x -= floorf(pos.x);
	pos.y -= floorf(pos.y);
	pos.z -= floorf(pos.z);

	// adjust for fade
	float u = fade(pos.x);
	float v = fade(pos.y);
	float w = fade(pos.z);

	// influence values
	float i000 = grad(randomIntGrid(ix, iy, iz, seed), pos.x, pos.y, pos.z);
	float i100 = grad(randomIntGrid(ix + 1, iy, iz, seed), pos.x - 1.0f, pos.y, pos.z);
	float i010 = grad(randomIntGrid(ix, iy + 1, iz, seed), pos.x, pos.y - 1.0f, pos.z);
	float i110 = grad(randomIntGrid(ix + 1, iy + 1, iz, seed), pos.x - 1.0f, pos.y - 1.0f, pos.z);
	float i001 = grad(randomIntGrid(ix, iy, iz + 1, seed), pos.x, pos.y, pos.z - 1.0f);
	float i101 = grad(randomIntGrid(ix + 1, iy, iz + 1, seed), pos.x - 1.0f, pos.y, pos.z - 1.0f);
	float i011 = grad(randomIntGrid(ix, iy + 1, iz + 1, seed), pos.x, pos.y - 1.0f, pos.z - 1.0f);
	float i111 = grad(randomIntGrid(ix + 1, iy + 1, iz + 1, seed), pos.x - 1.0f, pos.y - 1.0f, pos.z - 1.0f);

	// interpolation
	float x00 = lerp(i000, i100, u);
	float x10 = lerp(i010, i110, u);
	float x01 = lerp(i001, i101, u);
	float x11 = lerp(i011, i111, u);

	float y0 = lerp(x00, x10, v);
	float y1 = lerp(x01, x11, v);

	float avg = lerp(y0, y1, w);

	return avg;
}

__device__ float repeater(float3 pos, float scale, int seed, int n, float lacunarity, float decay, basisFunction basis)
{
	float acc = 0.0f;
	float amp = 1.0f;

	for (int i = 0; i < n; i++)
	{
		switch (basis)
		{
		case(CUDANOISE_CHECKER):
			acc += checker(make_float3(pos.x * scale, pos.y * scale, pos.z * scale), 1.0f, seed) * amp;
			break;
		case(CUDANOISE_LINEARVALUE):
			acc += linearValue(make_float3(pos.x * scale, pos.y * scale, pos.z * scale), 1.0f, seed) * amp;
			break;
		case(CUDANOISE_FADEDVALUE):
			acc += fadedValue(make_float3(pos.x * scale, pos.y * scale, pos.z * scale), 1.0f, seed) * amp;
			break;
		case(CUDANOISE_CUBICVALUE):
			acc += cubicValue(make_float3(pos.x * scale, pos.y * scale, pos.z * scale), 1.0f, seed) * amp;
			break;
		case(CUDANOISE_PERLIN):
			acc += perlinNoise(make_float3(pos.x * scale, pos.y * scale, pos.z * scale), 1.0f, seed) * amp;
			break;
		}

		scale *= lacunarity;
		amp *= decay;
	}

	return acc;
}

__device__ float turbulence(float3 pos, float scaleIn, float scaleOut, int seed, float strength, basisFunction inFunc, basisFunction outFunc)
{
	switch (inFunc)
	{
	case(CUDANOISE_CHECKER):
		pos.x += checker(pos, scaleIn, seed) * strength;
		pos.y += checker(pos, scaleIn, seed) * strength;
		pos.z += checker(pos, scaleIn, seed) * strength;
		break;
	case(CUDANOISE_LINEARVALUE):
		pos.x += linearValue(pos, scaleIn, seed) * strength;
		pos.y += linearValue(pos, scaleIn, seed) * strength;
		pos.z += linearValue(pos, scaleIn, seed) * strength;
		break;
	case(CUDANOISE_FADEDVALUE):
		pos.x += fadedValue(pos, scaleIn, seed) * strength;
		pos.y += fadedValue(pos, scaleIn, seed) * strength;
		pos.z += fadedValue(pos, scaleIn, seed) * strength;
		break;
	case(CUDANOISE_CUBICVALUE):
		pos.x += cubicValue(pos, scaleIn, seed) * strength;
		pos.y += cubicValue(pos, scaleIn, seed) * strength;
		pos.z += cubicValue(pos, scaleIn, seed) * strength;
		break;
	case(CUDANOISE_PERLIN):
		pos.x += perlinNoise(pos, scaleIn, seed) * strength;
		pos.y += perlinNoise(pos, scaleIn, seed) * strength;
		pos.z += perlinNoise(pos, scaleIn, seed) * strength;
		break;
	}

	switch (outFunc)
	{
	case(CUDANOISE_CHECKER):
		return checker(pos, scaleOut, seed);
		break;
	case(CUDANOISE_LINEARVALUE):
		return linearValue(pos, scaleOut, seed);
		break;
	case(CUDANOISE_FADEDVALUE):
		return fadedValue(pos, scaleOut, seed);
		break;
	case(CUDANOISE_CUBICVALUE):
		return cubicValue(pos, scaleOut, seed);
		break;
	case(CUDANOISE_PERLIN):
		return perlinNoise(pos, scaleOut, seed);
		break;
	}

	return 0.0f;
}

__device__ float repeaterTurbulence(float3 pos, float scaleIn, float scaleOut, int seed, float strength, int n, basisFunction basisIn, basisFunction basisOut)
{
	pos.x += (repeater(make_float3(pos.x, pos.y, pos.z), scaleIn, seed, n, 2.0f, 0.5f, basisIn)) * strength;

	return repeater(pos, scaleOut, seed, n, 2.0f, 0.75f, basisOut);
}

__device__ float recursiveTurbulence(float3 pos, int n, float harmonic, float decay, float strength)
{
	float3 displace = make_float3(pos.x, pos.y, pos.z);
	float scale = 1.0f;
	float amp = 1.0f;
	float acc = 0.0f;

	for (int i = 0; i < n; i++)
	{
		acc += perlinNoise(scaleVector(displace, scale), 1.0f, 0) * amp;

		displace.x += perlinNoise(make_float3(pos.x, pos.y, pos.z), 1.0f, 0) * amp * strength;
		displace.y += acc * strength;
		displace.z += perlinNoise(make_float3(acc, acc, acc), 1.0f, 0) * amp * strength;

		scale *= harmonic;
		amp *= decay;
	}

	return acc / 1.0f;
}
