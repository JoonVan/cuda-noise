#include "hip/hip_runtime.h"
// cudanoise
// Library of common 3D noise functions for CUDA kernels

#define N 512
#define WORLDSIZE N * N

#include <hip/hip_runtime.h>
#include "cudanoise.cuh"

__device__ unsigned int hash(unsigned int a)
{
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

__device__ float getRandomValue(unsigned int seed)
{
	unsigned int noiseVal = hash(seed);
	return ((float)noiseVal / (float)0xffffffff);
}

__device__ float clamp(float val, float min, float max)
{
	if (val < 0.0f)
		return 0.0f;
	else if (val > 1.0f)
		return 1.0f;

	return val;
}

__device__ float mapToSigned(float input)
{
	return input * 2.0f - 1.0f;
}

__device__ float mapToUnsigned(float input)
{
	return input * 0.5f + 0.5f;
}

__device__ float checker(float x, float y, float z, float scale)
{
	int ix = (int)(x * scale);
	int iy = (int)(y * scale);
	int iz = (int)(z * scale);

	if ((ix + iy + iz) % 2 == 0)
		return 1.0f;

	return 0.0f;
}

__device__ float rn(int x, int y, int z)
{
	return getRandomValue(x * 231 + y * 91023 + z * 48172);
}

__device__ float3 vectorNoise(int x, int y, int z)
{
	return make_float3(getRandomValue(x * 8231 + y * 34612 + z * 11836 + 19283) * 2.0f - 1.0f,
		   			   getRandomValue(x * 1171 + y * 9234 + z * 992903 + 1466) * 2.0f - 1.0f,
					   0.0f);
}

__device__ float3 scaleVector(float3 v, float factor)
{
	return make_float3(v.x * factor, v.y * factor, v.z * factor);
}

__device__ float3 addVectors(float3 v, float3 w)
{
	return make_float3(v.x + w.x, v.y + w.y, v.z + w.z);
}

__device__ float dotProduct(float3 u, float3 v)
{
	return (u.x * v.x + u.y * v.y + u.z * v.z);
}

__device__ float lerp(float a, float b, float ratio)
{
	return a * (1.0f - ratio) + b * ratio;
}

__device__ float cubic(float p0, float p1, float p2, float p3, float x)
{
	return p1 + 0.5 * x * (p2 - p0 + x * (2.0 * p0 - 5.0 * p1 + 4.0 * p2 - p3 + x * (3.0 * (p1 - p2) + p3 - p0)));
}

__device__ float tricubic(int x, int y, int z, float u, float v, float w)
{
	// interpolate along x first
	float x00 = cubic(rn(x - 1, y - 1, z - 1), rn(x, y - 1, z - 1), rn(x + 1, y - 1, z - 1), rn(x + 2, y - 1, z - 1), u);
	float x01 = cubic(rn(x - 1, y - 1, z), rn(x, y - 1, z), rn(x + 1, y - 1, z), rn(x + 2, y - 1, z), u);
	float x02 = cubic(rn(x - 1, y - 1, z + 1), rn(x, y - 1, z + 1), rn(x + 1, y - 1, z + 1), rn(x + 2, y - 1, z + 1), u);
	float x03 = cubic(rn(x - 1, y - 1, z + 2), rn(x, y - 1, z + 2), rn(x + 1, y - 1, z + 2), rn(x + 2, y - 1, z + 2), u);

	float x10 = cubic(rn(x - 1, y, z - 1), rn(x, y, z - 1), rn(x + 1, y, z - 1), rn(x + 2, y, z - 1), u);
	float x11 = cubic(rn(x - 1, y, z), rn(x, y, z), rn(x + 1, y, z), rn(x + 2, y, z), u);
	float x12 = cubic(rn(x - 1, y, z + 1), rn(x, y, z + 1), rn(x + 1, y, z + 1), rn(x + 2, y, z + 1), u);
	float x13 = cubic(rn(x - 1, y, z + 2), rn(x, y, z + 2), rn(x + 1, y, z + 2), rn(x + 2, y, z + 2), u);

	float x20 = cubic(rn(x - 1, y + 1, z - 1), rn(x, y + 1, z - 1), rn(x + 1, y + 1, z - 1), rn(x + 2, y + 1, z - 1), u);
	float x21 = cubic(rn(x - 1, y + 1, z), rn(x, y + 1, z), rn(x + 1, y + 1, z), rn(x + 2, y + 1, z), u);
	float x22 = cubic(rn(x - 1, y + 1, z + 1), rn(x, y + 1, z + 1), rn(x + 1, y + 1, z + 1), rn(x + 2, y + 1, z + 1), u);
	float x23 = cubic(rn(x - 1, y + 1, z + 2), rn(x, y + 1, z + 2), rn(x + 1, y + 1, z + 2), rn(x + 2, y + 1, z + 2), u);

	float x30 = cubic(rn(x - 1, y + 2, z - 1), rn(x, y + 2, z - 1), rn(x + 1, y + 2, z - 1), rn(x + 2, y + 2, z - 1), u);
	float x31 = cubic(rn(x - 1, y + 2, z), rn(x, y + 2, z), rn(x + 1, y + 2, z), rn(x + 2, y + 2, z), u);
	float x32 = cubic(rn(x - 1, y + 2, z + 1), rn(x, y + 2, z + 1), rn(x + 1, y + 2, z + 1), rn(x + 2, y + 2, z + 1), u);
	float x33 = cubic(rn(x - 1, y + 2, z + 2), rn(x, y + 2, z + 2), rn(x + 1, y + 2, z + 2), rn(x + 2, y + 2, z + 2), u);

	// interpolate along y
	float y0 = cubic(x00, x10, x20, x30, v);
	float y1 = cubic(x01, x11, x21, x31, v);
	float y2 = cubic(x02, x12, x22, x32, v);
	float y3 = cubic(x03, x13, x23, x33, v);

	// interpolate along z
	return cubic(y0, y1, y2, y3, w);

}

__device__ float discreteNoise(float x, float y, float z, float scale)
{
	int ix = (int)(x * scale);
	int iy = (int)(y * scale);
	int iz = (int)(z * scale);

	return rn(ix, iy, iz);
}

__device__ float cubicValue(float3 pos, float scale = 1.0f)
{
	int ix = (int)(pos.x * scale);
	int iy = (int)(pos.y * scale);
	int iz = (int)(pos.z * scale);

	float u = pos.x - ix;
	float v = pos.y - iy;
	float w = pos.z - iz;

	return tricubic(ix, iy, iz, u, v, w);
}

__device__ float linearValue(float3 pos, float scale = 1.0f)
{
	int ix = (int)(pos.x * scale);
	int iy = (int)(pos.y * scale);
	int iz = (int)(pos.z * scale);

	float u = pos.x - ix;
	float v = pos.y - iy;
	float w = pos.z - iz;

	// Corner values
	float a000 = rn(ix, iy, iz);
	float a100 = rn(ix + 1, iy, iz);
	float a010 = rn(ix, iy + 1, iz);
	float a110 = rn(ix + 1, iy + 1, iz);
	float a001 = rn(ix, iy, iz + 1);
	float a101 = rn(ix + 1, iy, iz + 1);
	float a011 = rn(ix, iy + 1, iz + 1);
	float a111 = rn(ix + 1, iy + 1, iz + 1);

	// Linear interpolation
	float x00 = lerp(a000, a100, u);
	float x10 = lerp(a010, a110, u);
	float x01 = lerp(a001, a101, u);
	float x11 = lerp(a011, a111, u);

	float y0 = lerp(x00, x10, v);
	float y1 = lerp(x01, x11, v);

	return lerp(y0, y1, w) / 2.0f * 1.0f;
}

__device__ float grad(int hash, float x, float y, float z)
{
	switch (hash & 0xF)
	{
		case 0x0: return  x + y;
		case 0x1: return -x + y;
		case 0x2: return  x - y;
		case 0x3: return -x - y;
		case 0x4: return  x + z;
		case 0x5: return -x + z;
		case 0x6: return  x - z;
		case 0x7: return -x - z;
		case 0x8: return  y + z;
		case 0x9: return -y + z;
		case 0xA: return  y - z;
		case 0xB: return -y - z;
		case 0xC: return  y + x;
		case 0xD: return -y + z;
		case 0xE: return  y - x;
		case 0xF: return -y - z;
	default: return 0; // never happens
	}
}

__device__ int getHash(int x, int y, int z)
{
	return hash((unsigned int)(x * 1723 + y * 93241 + z * 149812 + 3824));
}

__device__ float fade(float t) 
{
	// Fade function as defined by Ken Perlin.  This eases coordinate values
	// so that they will ease towards integral values.  This ends up smoothing
	// the final output.
	return t * t * t * (t * (t * 6.0f - 15.0f) + 10.0f);         // 6t^5 - 15t^4 + 10t^3
}

__device__ float perlinNoise(float3 pos)
{
	// zero corner integer position
	int ix = (int)floorf(pos.x);
	int iy = (int)floorf(pos.y);
	int iz = (int)floorf(pos.z);

	// current position within unit cube
	pos.x -= floorf(pos.x);
	pos.y -= floorf(pos.y);
	pos.z -= floorf(pos.z);

	// adjust for fade
	float u = fade(pos.x);
	float v = fade(pos.y);
	float w = fade(pos.z);

	// influence values
	float i000 = grad(getHash(ix, iy, iz), pos.x, pos.y, pos.z);
	float i100 = grad(getHash(ix + 1, iy, iz), pos.x - 1.0f, pos.y, pos.z);
	float i010 = grad(getHash(ix, iy + 1, iz), pos.x, pos.y - 1.0f, pos.z);
	float i110 = grad(getHash(ix + 1, iy + 1, iz), pos.x - 1.0f, pos.y - 1.0f, pos.z);
	float i001 = grad(getHash(ix, iy, iz + 1), pos.x, pos.y, pos.z - 1.0f);
	float i101 = grad(getHash(ix + 1, iy, iz + 1), pos.x - 1.0f, pos.y, pos.z - 1.0f);
	float i011 = grad(getHash(ix, iy + 1, iz + 1), pos.x, pos.y - 1.0f, pos.z - 1.0f);
	float i111 = grad(getHash(ix + 1, iy + 1, iz + 1), pos.x - 1.0f, pos.y - 1.0f, pos.z - 1.0f);

	// interpolation
	float x00 = lerp(i000, i100, u);
	float x10 = lerp(i010, i110, u);
	float x01 = lerp(i001, i101, u);
	float x11 = lerp(i011, i111, u);

	float y0 = lerp(x00, x10, v);
	float y1 = lerp(x01, x11, v);

	float avg = lerp(y0, y1, w);

	return avg;
}

__device__ float repeater(float3 pos, int n, float harmonic = 2.0f, float decay = 0.5f, basisFunction basis = CUDANOISE_PERLIN)
{
	float scale = 1.0f;
	float acc = 0.0f;
	float amp = 1.0f;

	for (int i = 0; i < n; i++)
	{
		switch (basis)
		{
		case(CUDANOISE_LINEARVALUE):
			acc += linearValue(make_float3(pos.x * scale, pos.y * scale, pos.z * scale)) * amp;
			break;
		case(CUDANOISE_CUBICVALUE):
			acc += cubicValue(make_float3(pos.x * scale, pos.y * scale, pos.z * scale)) * amp;
			break;
		case(CUDANOISE_PERLIN):
			acc += perlinNoise(make_float3(pos.x * scale, pos.y * scale, pos.z * scale)) * amp;
			break;
		}

		scale *= harmonic;
		amp *= decay;
	}

	return acc;
}

__device__ float turbulence(float3 pos, float strength)
{
	pos.x += perlinNoise(pos) * strength;

	return perlinNoise(pos);
}

__device__ float repeaterTurbulence(float3 pos, float strength, int n)
{
	pos.x += (repeater(make_float3(pos.x, pos.y, pos.z), n)) * strength;

	return repeater(pos, n);
}

__device__ float recursiveTurbulence(float3 pos, int n, float harmonic = 2.0f, float decay = 0.5f, float strength = 1.0f)
{
	float3 displace = make_float3(pos.x, pos.y, pos.z);
	float scale = 1.0f;
	float amp = 1.0f;
	float acc = 0.0f;

	for (int i = 0; i < n; i++)
	{
		acc += perlinNoise(scaleVector(displace, scale)) * amp;

		displace.x += perlinNoise(make_float3(pos.x, pos.y, pos.z)) * amp * strength;
		displace.y += acc * strength;
		displace.z += perlinNoise(make_float3(acc, acc, acc)) * amp * strength;

		scale *= harmonic;
		amp *= decay;
	}

	return acc / 1.0f;
}

__device__ float recursiveRepeaterTurbulence(float3 pos, int n, int m, float harmonic = 2.0f, float decay = 0.5f, float strength = 1.0f)
{
	float3 displace = make_float3(pos.x, pos.y, pos.z);
	float scale = 1.0f;
	float amp = 1.0f;
	float acc = 0.0f;

	for (int i = 0; i < n; i++)
	{
		acc += repeater(scaleVector(displace, scale), m) * amp;

		displace.x += repeater(make_float3(pos.x, pos.y, pos.z), m) * amp * strength;
		displace.y += acc * strength;
		displace.z += repeater(make_float3(acc, acc, acc), m) * amp * strength;

		scale *= harmonic;
		amp *= decay;
	}

	return acc / 1.0f;
}